#include "hip/hip_runtime.h"
/** Copyright (c) 2022 NVIDIA CORPORATION.  All rights reserved.
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

#include "warp.h"
#include "cuda_util.h"

#include <hip/hiprtc.h>

#include <map>
#include <vector>

struct DeviceInfo
{
    static constexpr int kNameLen = 128;

    hipDevice_t device = -1;
    int ordinal = -1;
    char name[kNameLen] = "";
    int arch = 0;
    int is_uva = 0;
};

struct ContextInfo
{
    DeviceInfo* device_info = NULL;

    hipStream_t stream = NULL; // created when needed
};

// cached info for all devices, indexed by ordinal
static std::vector<DeviceInfo> g_devices;

// maps hipDevice_t to DeviceInfo
static std::map<hipDevice_t, DeviceInfo*> g_device_map;

// cached info for all known contexts
static std::map<hipCtx_t, ContextInfo> g_contexts;


void cuda_set_context_restore_policy(bool always_restore)
{
    ContextGuard::always_restore = always_restore;
}

int cuda_get_context_restore_policy()
{
    return int(ContextGuard::always_restore);
}

int cuda_init()
{
    if (!init_cuda_driver())
        return -1;

    int deviceCount = 0;
    if (check_cu(cuDeviceGetCount_f(&deviceCount)))
    {
        g_devices.resize(deviceCount);

        for (int i = 0; i < deviceCount; i++)
        {
            hipDevice_t device;
            if (check_cu(cuDeviceGet_f(&device, i)))
            {
                // query device info
                g_devices[i].device = device;
                g_devices[i].ordinal = i;
                check_cu(cuDeviceGetName_f(g_devices[i].name, DeviceInfo::kNameLen, device));
                check_cu(cuDeviceGetAttribute_f(&g_devices[i].is_uva, hipDeviceAttributeUnifiedAddressing, device));
                int major = 0;
                int minor = 0;
                check_cu(cuDeviceGetAttribute_f(&major, hipDeviceAttributeComputeCapabilityMajor, device));
                check_cu(cuDeviceGetAttribute_f(&minor, hipDeviceAttributeComputeCapabilityMinor, device));
                g_devices[i].arch = 10 * major + minor;

                g_device_map[device] = &g_devices[i];
            }
            else
            {
                return -1;
            }
        }
    }
    else
    {
        return -1;
    }

    return 0;
}


static inline hipCtx_t get_current_context()
{
    hipCtx_t ctx;
    if (check_cu(cuCtxGetCurrent_f(&ctx)))
        return ctx;
    else
        return NULL;
}

static inline hipStream_t get_current_stream()
{
    return static_cast<hipStream_t>(cuda_context_get_stream(NULL));
}

static ContextInfo* get_context_info(hipCtx_t ctx)
{
    if (!ctx)
    {
        ctx = get_current_context();
        if (!ctx)
            return NULL;
    }

    auto it = g_contexts.find(ctx);
    if (it != g_contexts.end())
    {
        return &it->second;
    }
    else
    {
        // previously unseen context, add the info
        ContextGuard guard(ctx, true);
        ContextInfo context_info;
        hipDevice_t device;
        if (check_cu(cuCtxGetDevice_f(&device)))
        {
            context_info.device_info = g_device_map[device];
            auto result = g_contexts.insert(std::make_pair(ctx, context_info));
            return &result.first->second;
        }
    }

    return NULL;
}


// void* alloc_host(size_t s)
// {
//     void* ptr;
//     check_cuda(hipHostMalloc(&ptr, s));
//     return ptr;
// }

// void free_host(void* ptr)
// {
//     hipHostFree(ptr);
// }

void* alloc_device(void* context, size_t s)
{
    ContextGuard guard(context);

    void* ptr;
    check_cuda(hipMalloc(&ptr, s));
    return ptr;
}

void free_device(void* context, void* ptr)
{
    ContextGuard guard(context);

    check_cuda(hipFree(ptr));
}

void memcpy_h2d(void* context, void* dest, void* src, size_t n)
{
    ContextGuard guard(context);
    
    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyHostToDevice, get_current_stream()));
}

void memcpy_d2h(void* context, void* dest, void* src, size_t n)
{
    ContextGuard guard(context);

    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToHost, get_current_stream()));
}

void memcpy_d2d(void* context, void* dest, void* src, size_t n)
{
    ContextGuard guard(context);

    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToDevice, get_current_stream()));
}

void memcpy_peer(void* dest_context, void* dest, void* src_context, void* src, size_t n)
{
    ContextGuard guard(dest_context);

    check_cu(cuMemcpyPeerAsync_f(
        reinterpret_cast<hipDeviceptr_t>(dest), static_cast<hipCtx_t>(dest_context),
        reinterpret_cast<hipDeviceptr_t>(src), static_cast<hipCtx_t>(src_context),
        n, get_current_stream()
    ));
}

__global__ void memset_kernel(int* dest, int value, int n)
{
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (tid < n)
    {
        dest[tid] = value;
    }
}

void memset_device(void* context, void* dest, int value, size_t n)
{
    ContextGuard guard(context);

    if ((n%4) > 0)
    {
        // for unaligned lengths fallback to CUDA memset
        check_cuda(hipMemsetAsync(dest, value, n, get_current_stream()));
    }
    else
    {
        // custom kernel to support 4-byte values (and slightly lower host overhead)
        const int num_words = n/4;
        wp_launch_device(WP_CURRENT_CONTEXT, memset_kernel, num_words, ((int*)dest, value, num_words));
    }
}


void array_inner_device(uint64_t a, uint64_t b, uint64_t out, int len)
{

}

void array_sum_device(uint64_t a, uint64_t out, int len)
{
    
}


int cuda_device_get_count()
{
    int count = 0;
    check_cu(cuDeviceGetCount_f(&count));
    return count;
}

void* cuda_device_primary_context_retain(int ordinal)
{
    hipCtx_t context = NULL;
    hipDevice_t device;
    if (check_cu(cuDeviceGet_f(&device, ordinal)))
        check_cu(cuDevicePrimaryCtxRetain_f(&context, device));
    return context;
}

void cuda_device_primary_context_release(int ordinal)
{
    hipDevice_t device;
    if (check_cu(cuDeviceGet_f(&device, ordinal)))
        check_cu(cuDevicePrimaryCtxRelease_f(device));
}

const char* cuda_device_get_name(int ordinal)
{
    if (ordinal >= 0 && ordinal < int(g_devices.size()))
        return g_devices[ordinal].name;
    return NULL;
}

int cuda_device_get_arch(int ordinal)
{
    if (ordinal >= 0 && ordinal < int(g_devices.size()))
        return g_devices[ordinal].arch;
    return 0;
}

int cuda_device_is_uva(int ordinal)
{
    if (ordinal >= 0 && ordinal < int(g_devices.size()))
        return g_devices[ordinal].is_uva;
    return 0;
}

void* cuda_context_get_current()
{
    return get_current_context();
}

void cuda_context_set_current(void* context)
{
    hipCtx_t ctx = static_cast<hipCtx_t>(context);
    hipCtx_t prev_ctx = NULL;
    check_cu(cuCtxGetCurrent_f(&prev_ctx));
    if (ctx != prev_ctx)
    {
        check_cu(cuCtxSetCurrent_f(ctx));
    }
}

void cuda_context_push_current(void* context)
{
    check_cu(cuCtxPushCurrent_f(static_cast<hipCtx_t>(context)));
}

void cuda_context_pop_current()
{
    hipCtx_t context;
    check_cu(cuCtxPopCurrent_f(&context));
}

void* cuda_context_create(int device_ordinal)
{
    hipCtx_t ctx = NULL;
    hipDevice_t device;
    if (check_cu(cuDeviceGet_f(&device, device_ordinal)))
        check_cu(cuCtxCreate_f(&ctx, 0, device));
    return ctx;
}

void cuda_context_destroy(void* context)
{
    if (context)
    {
        hipCtx_t ctx = static_cast<hipCtx_t>(context);

        // ensure this is not the current context
        if (ctx == cuda_context_get_current())
            cuda_context_set_current(NULL);

        // release the cached info about this context
        ContextInfo* info = get_context_info(ctx);
        if (info)
        {
            if (info->stream)
                check_cu(cuStreamDestroy_f(info->stream));
            
            g_contexts.erase(ctx);
        }

        check_cu(cuCtxDestroy_f(ctx));
    }
}

void cuda_context_synchronize(void* context)
{
    ContextGuard guard(context);

    check_cu(cuCtxSynchronize_f());
}

uint64_t cuda_context_check(void* context)
{
    ContextGuard guard(context);

    hipStreamCaptureStatus status;
    hipStreamIsCapturing(get_current_stream(), &status);
    
    // do not check during cuda stream capture
    // since we cannot synchronize the device
    if (status == hipStreamCaptureStatusNone)
    {
        hipDeviceSynchronize();
        return hipPeekAtLastError(); 
    }
    else
    {
        return 0;
    }
}


int cuda_context_get_device_ordinal(void* context)
{
    ContextInfo* info = get_context_info(static_cast<hipCtx_t>(context));
    return info && info->device_info ? info->device_info->ordinal : -1;
}

int cuda_context_is_primary(void* context)
{
    int ordinal = cuda_context_get_device_ordinal(context);
    if (ordinal != -1)
    {
        // there is no CUDA API to check if a context is primary, but we can temporarily
        // acquire the device's primary context to check the pointer
        void* device_primary_context = cuda_device_primary_context_retain(ordinal);
        cuda_device_primary_context_release(ordinal);
        return int(context == device_primary_context);
    }
    return 0;
}

void* cuda_context_get_stream(void* context)
{
    ContextInfo* info = get_context_info(static_cast<hipCtx_t>(context));
    if (info)
    {
        // create stream on demand
        if (!info->stream)
        {
            ContextGuard guard(context, true);
            check_cu(cuStreamCreate_f(&info->stream, hipStreamDefault));
        }
        return info->stream;
    }
    return NULL;
}

int cuda_context_enable_peer_access(void* context, void* peer_context)
{
    if (!context || !peer_context)
    {
        fprintf(stderr, "Warp error: Failed to enable peer access: invalid argument\n");
        return 0;
    }

    if (context == peer_context)
        return 1;  // ok

    hipCtx_t ctx = static_cast<hipCtx_t>(context);
    hipCtx_t peer_ctx = static_cast<hipCtx_t>(peer_context);

    ContextInfo* info = get_context_info(ctx);
    ContextInfo* peer_info = get_context_info(peer_ctx);
    if (!info || !peer_info)
    {
        fprintf(stderr, "Warp error: Failed to enable peer access: failed to get context info\n");
        return 0;
    }

    // check if same device
    if (info->device_info == peer_info->device_info)
    {
        if (info->device_info->is_uva)
        {
            return 1;  // ok
        }
        else
        {
            fprintf(stderr, "Warp error: Failed to enable peer access: device doesn't support UVA\n");
            return 0;
        }
    }
    else
    {
        // different devices, try to enable
        ContextGuard guard(ctx, true);
        hipError_t result = cuCtxEnablePeerAccess_f(peer_ctx, 0);
        if (result == hipSuccess || result == hipErrorPeerAccessAlreadyEnabled)
        {
            return 1;  // ok
        }
        else
        {
            check_cu(result);
            return 0;
        }
    }
}

int cuda_context_can_access_peer(void* context, void* peer_context)
{
    if (!context || !peer_context)
        return 0;

    if (context == peer_context)
        return 1;

    hipCtx_t ctx = static_cast<hipCtx_t>(context);
    hipCtx_t peer_ctx = static_cast<hipCtx_t>(peer_context);
    
    ContextInfo* info = get_context_info(ctx);
    ContextInfo* peer_info = get_context_info(peer_ctx);
    if (!info || !peer_info)
        return 0;

    // check if same device
    if (info->device_info == peer_info->device_info)
    {
        if (info->device_info->is_uva)
            return 1;
        else
            return 0;
    }
    else
    {
        // different devices, try to enable
        // TODO: is there a better way to check?
        ContextGuard guard(ctx, true);
        hipError_t result = cuCtxEnablePeerAccess_f(peer_ctx, 0);
        if (result == hipSuccess || result == hipErrorPeerAccessAlreadyEnabled)
            return 1;
        else
            return 0;
    }
}

void* cuda_stream_get_current()
{
    return get_current_stream();
}

void cuda_graph_begin_capture(void* context)
{
    ContextGuard guard(context);

    check_cuda(hipStreamBeginCapture(get_current_stream(), hipStreamCaptureModeGlobal));
}

void* cuda_graph_end_capture(void* context)
{
    ContextGuard guard(context);

    hipGraph_t graph = NULL;
    check_cuda(hipStreamEndCapture(get_current_stream(), &graph));

    if (graph)
    {
        // enable to create debug GraphVis visualization of graph
        //hipGraphDebugDotPrint(graph, "graph.dot", hipGraphDebugDotFlagsVerbose);

        hipGraphExec_t graph_exec = NULL;
        check_cuda(hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0));
        
        // can use after CUDA 11.4 to permit graphs to capture hipMallocAsync() operations
        //check_cuda(hipGraphInstantiateWithFlags(&graph_exec, graph, hipGraphInstantiateFlagAutoFreeOnLaunch));

        // free source graph
        check_cuda(hipGraphDestroy(graph));

        return graph_exec;
    }
    else
    {
        return NULL;
    }
}

void cuda_graph_launch(void* context, void* graph_exec)
{
    ContextGuard guard(context);

    check_cuda(hipGraphLaunch((hipGraphExec_t)graph_exec, get_current_stream()));
}

void cuda_graph_destroy(void* context, void* graph_exec)
{
    ContextGuard guard(context);

    check_cuda(hipGraphExecDestroy((hipGraphExec_t)graph_exec));
}

size_t cuda_compile_program(const char* cuda_src, int arch, const char* include_dir, bool debug, bool verbose, bool verify_fp, const char* output_file)
{
    hiprtcResult res;

    hiprtcProgram prog;
    res = hiprtcCreateProgram(
        &prog,          // prog
        cuda_src,      // buffer
        NULL,          // name
        0,             // numHeaders
        NULL,          // headers
        NULL);         // includeNames

    if (res != HIPRTC_SUCCESS)
        return res;

    // check include dir path len (path + option)
    const int max_path = 4096 + 16;
    if (strlen(include_dir) > max_path)
    {
        printf("Include path too long\n");
        return size_t(-1);
    }

    char include_opt[max_path];
    strcpy(include_opt, "--include-path=");
    strcat(include_opt, include_dir);

    const int max_arch = 256;
    char arch_opt[max_arch];
    sprintf(arch_opt, "--gpu-architecture=compute_%d", arch);

    const char *opts[] = 
    {
        "--device-as-default-execution-space",
        arch_opt,
        "--use_fast_math",
        "--std=c++11",
        "--define-macro=WP_CUDA",
        (verify_fp ? "--define-macro=WP_VERIFY_FP" : "--undefine-macro=WP_VERIFY_FP"),
        "--define-macro=WP_NO_CRT",
        (debug ? "--define-macro=DEBUG" : "--define-macro=NDEBUG"),
        include_opt
    };

    res = hiprtcCompileProgram(prog, 9, opts);

    if (res == HIPRTC_SUCCESS)
    {
        // save ptx
        size_t ptx_size;
        hiprtcGetCodeSize(prog, &ptx_size);

        char* ptx = (char*)malloc(ptx_size);
        hiprtcGetCode(prog, ptx);

        // write to file
        FILE* file = fopen(output_file, "w");
        fwrite(ptx, 1, ptx_size, file);
        fclose(file);

        free(ptx);
    }

    if (res != HIPRTC_SUCCESS || verbose)
    {
        // get program log
        size_t log_size;
        hiprtcGetProgramLogSize(prog, &log_size);

        char* log = (char*)malloc(log_size);
        hiprtcGetProgramLog(prog, log);

        // todo: figure out better way to return this to python
        printf("%s", log);
        free(log);
    }

    hiprtcDestroyProgram(&prog);
    return res;
}

void* cuda_load_module(void* context, const char* path)
{
    ContextGuard guard(context);

    FILE* file = fopen(path, "rb");
    fseek(file, 0, SEEK_END);
    size_t length = ftell(file);
    fseek(file, 0, SEEK_SET);

    char* buf = (char*)malloc(length);
    size_t result = fread(buf, 1, length, file);
    fclose(file);

    if (result != length)
    {
        printf("Warp: Failed to load PTX from disk, unexpected number of bytes\n");
        return NULL;
    }

    hipJitOption options[2];
    void *optionVals[2];
    char error_log[8192];
    unsigned int logSize = 8192;
    // Setup linker options
    // Pass a buffer for error message
    options[0] = hipJitOptionErrorLogBuffer;
    optionVals[0] = (void *) error_log;
    // Pass the size of the error buffer
    options[1] = hipJitOptionErrorLogBufferSizeBytes;
    optionVals[1] = (void *) (long) logSize;

    hipModule_t module = NULL;
    if (!check_cu(cuModuleLoadDataEx_f(&module, buf, 2, options, optionVals)))
    {
        printf("Warp: Loading PTX module failed\n");
        // print error log
        fprintf(stderr, "PTX linker error:\n%s\n", error_log);
    }

    free(buf);

    return module;
}

void cuda_unload_module(void* context, void* module)
{
    ContextGuard guard(context);

    check_cu(cuModuleUnload_f((hipModule_t)module));
}

void* cuda_get_kernel(void* context, void* module, const char* name)
{
    ContextGuard guard(context);

    hipFunction_t kernel = NULL;
    if (!check_cu(cuModuleGetFunction_f(&kernel, (hipModule_t)module, name)))
        printf("Warp: Failed to lookup kernel function %s in module\n", name);

    return kernel;
}

size_t cuda_launch_kernel(void* context, void* kernel, size_t dim, void** args)
{
    ContextGuard guard(context);

    const int block_dim = 256;
    const int grid_dim = (dim + block_dim - 1)/block_dim;

    hipError_t res = cuLaunchKernel_f(
        (hipFunction_t)kernel,
        grid_dim, 1, 1,
        block_dim, 1, 1,
        0, get_current_stream(),
        args,
        0);

    return res;

}

// impl. files
#include "bvh.cu"
#include "mesh.cu"
#include "sort.cu"
#include "hashgrid.cu"
#include "marching.cu"
#include "volume_builder.cu"

//#include "spline.inl"
//#include "volume.inl"

